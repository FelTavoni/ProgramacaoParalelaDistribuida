#include "hip/hip_runtime.h"
/*
* Paralelized marathon code - CUDA mapped.
* 
* Strategy: Each core will calculate one position of the heating cube!
*
* Universidade Federal de São Carlos,
* Felipe Tavoni.
*/

#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <string.h>
#include <math.h>
#define STABILITY 1.0f/sqrt(3.0f)


__global__ void mdf_heat_once(double*  __restrict__ u0, 
								double*  __restrict__ u1, 
								const unsigned int* npX, 
								const unsigned int* npY, 
								const unsigned int* npZ,
								const double* deltaH,
								const double* deltaT,
								const double* alpha,
								const double* inErr,
								const double* boundaries){

	const unsigned int z = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int x = blockIdx.z * blockDim.z + threadIdx.z;

	// 3d flattened matrix:
	// x + (width * y) + (width * height * z)
	// which is: z + ((*npZ) * y) + ((*npZ) * (*npY) * x)

	double left   = *boundaries;
	double right  = *boundaries;
	double up     = *boundaries;
	double down   = *boundaries;
	double top    = *boundaries;
	double bottom = *boundaries;

	if ((z < (*npZ)) && (y < (*npY)) && (x < (*npX))) {

		if ((x > 0) && (x < ((*npX) - 1))){
			left  = u0[z + ((*npZ) * y) + ((*npZ) * (*npY) * (x-1))];
			right = u0[z + ((*npZ) * y) + ((*npZ) * (*npY) * (x+1))];
		} else if (x == 0) 
			right = u0[z + ((*npZ) * y) + ((*npZ) * (*npY) * (x+1))];
		else 
			left = u0[z + ((*npZ) * y) + ((*npZ) * (*npY) * (x-1))];

		if ((y > 0) && (y < ((*npY) - 1))){
			up  = u0[z + ((*npZ) * (y-1)) + ((*npZ) * (*npY) * x)];
			down = u0[z + ((*npZ) * (y+1)) + ((*npZ) * (*npY) * x)];
		} else if (y == 0) 
			down = u0[z + ((*npZ) * (y+1)) + ((*npZ) * (*npY) * x)];
		else 
			up = u0[z + ((*npZ) * (y-1)) + ((*npZ) * (*npY) * x)];

		if ((z > 0) && (z < ((*npZ) - 1))){
			top  = u0[(z-1) + ((*npZ) * y) + ((*npZ) * (*npY) * x)];
			bottom = u0[(z+1) + ((*npZ) * y) + ((*npZ) * (*npY) * x)];
		} else if (z == 0) 
			bottom = u0[(z+1) + ((*npZ) * y) + ((*npZ) * (*npY) * x)];
		else 
			top = u0[(z-1) + ((*npZ) * y) + ((*npZ) * (*npY) * x)];

		// Simply applying the formula and stores the value on a new spot.
		u1[z + ((*npZ) * y) + ((*npZ) * (*npY) * x)] =  (*alpha) * ( top + bottom + up + down + left + right  - (6.0f * u0[z + ((*npZ) * y) + ((*npZ) * (*npY) * x)] )) + u0[z + ((*npZ) * y) + ((*npZ) * (*npY) * x)];

		// printf("(%d, %d, %d) is %lf\n", z, y, x, u1[z + ((*npZ) * y) + ((*npZ) * (*npY) * x)]);

	}

}

__global__ void mdf_heat_check(double*  __restrict__ u0, 
								double*  __restrict__ u1, 
								const unsigned int* npX, 
								const unsigned int* npY, 
								const unsigned int* npZ,
								const double* inErr,
								const double* boundaries,
								volatile int* heated){

	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;

	// If all the positions are heated more than 100, finish the iteration.
	double err = 0.0f;
	if ((z < (*npZ)) && (y < (*npY)) && (x < (*npX))) {
		err = fabs(u0[z + (*npY) * (y + (*npX) * x)] - (*boundaries));
		if (err <= (*inErr))
			*heated = 0;
	}
}

int onDevice(unsigned int h_npX, unsigned int h_npY, unsigned int h_npZ, double h_deltaH, double h_deltaT, double h_alpha, double h_boundaries, double h_inErr) {

	// Checking the GPUs available...
	int nDevices;
	hipGetDeviceCount(&nDevices);
  	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Memory Clock Rate (KHz): %d\n",
			prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n",
			prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
			2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  	}

	// Allocate variables in the GPU and copy they content from host.
	double *d_deltaT; //0.01;
	double *d_deltaH;  //0.25f;
	unsigned int *d_npX;  //1.0f;
	unsigned int *d_npY;  //1.0f;
	unsigned int *d_npZ;  //1.0f;
	// Constant variables.
	double *d_boundaries;
	double *d_inErr;
	double *d_alpha;

	hipError_t err;

	hipMalloc((void**)&d_deltaT, sizeof(double));
	hipMemcpy(d_deltaT, &h_deltaT, sizeof(double), hipMemcpyHostToDevice);
	hipMalloc((void**)&d_deltaH, sizeof(double));
	hipMemcpy(d_deltaH, &h_deltaH, sizeof(double), hipMemcpyHostToDevice);
	hipMalloc((void**)&d_npX, sizeof(double));
	hipMemcpy(d_npX, &h_npX, sizeof(double), hipMemcpyHostToDevice);
	hipMalloc((void**)&d_npY, sizeof(double));
	hipMemcpy(d_npY, &h_npY, sizeof(double), hipMemcpyHostToDevice);
	hipMalloc((void**)&d_npZ, sizeof(double));
	hipMemcpy(d_npZ, &h_npZ, sizeof(double), hipMemcpyHostToDevice);
	hipMalloc((void**)&d_boundaries, sizeof(double));
	hipMemcpy(d_boundaries, &h_boundaries, sizeof(double), hipMemcpyHostToDevice);
	hipMalloc((void**)&d_inErr, sizeof(double));
	hipMemcpy(d_inErr, &h_inErr, sizeof(double), hipMemcpyHostToDevice);
	hipMalloc((void**)&d_alpha, sizeof(double));
	hipMemcpy(d_alpha, &h_alpha, sizeof(double), hipMemcpyHostToDevice);

	// Allocate memory inside the GPU for the grid. Here the matrix is flattened.
	double *d_u0;
	double *d_u1;
	hipMalloc((void**)&d_u0, h_npZ * h_npY * h_npX * sizeof(double));
	hipMemset((void**)d_u0, 0x00, h_npZ * h_npY * h_npX * sizeof(double));
	hipMalloc((void**)&d_u1, h_npZ * h_npY * h_npX * sizeof(double));
	hipMemset((void**)d_u1, 0x00, h_npZ * h_npY * h_npX * sizeof(double));

	double steps = 0;
	int *heated = (int*)malloc(sizeof(int)); 
	(*heated) = 1;
	int *d_heated;
	hipMalloc((void**)&d_heated, sizeof(int));

	// Defining the grid.
	dim3 threadsPerBlock(4, 4, 4); // 4 * 4 * 4 = 64 threads = 2 warps!
	dim3 blocksPerGrid(ceil( (double)h_npZ/4), ceil( (double)h_npY/4), ceil( (double)h_npX/4));

	while (*heated) {

		steps++;
		
		// Calling the kernel for heat function.
		mdf_heat_once<<<blocksPerGrid, threadsPerBlock>>>(d_u0, d_u1, d_npX, d_npY, d_npZ, d_deltaH, d_deltaT, d_alpha, d_inErr, d_boundaries);
		hipDeviceSynchronize();

		err = hipGetLastError();
		if (err != hipSuccess) 
			printf("Error: %s\n", hipGetErrorString(err));

		// Switch the cubes, since the previous won't be reused, so we don't need to allocate more memory.
		double *ptr = d_u0;
		d_u0 = d_u1;
		d_u1 = ptr;

		// Let's assume the cube is heated. We'll assure that by checking every position searching if there's a spot that
		//	has not been heated enough.
		(*heated) = 1;
		hipMemcpy(d_heated, heated, sizeof(int), hipMemcpyHostToDevice);
		mdf_heat_check<<<blocksPerGrid, threadsPerBlock>>>(d_u0, d_u1, d_npX, d_npY, d_npZ, d_inErr, d_boundaries, d_heated);
		hipDeviceSynchronize();
		hipMemcpy(heated, d_heated, sizeof(int), hipMemcpyDeviceToHost);

	}

	printf("Steps: %.1lf\n", steps);

	// Free the space in GPU.
	return EXIT_SUCCESS;
}

int onHost() {
	// Define variables to be used in the process
	double h_deltaT = 0.0f; //0.01;
	double h_deltaH =0.0f;  //0.25f;
	double h_sizeX = 0.0f;  //1.0f;
	double h_sizeY = 0.0f;  //1.0f;
	double h_sizeZ = 0.0f;  //1.0f;

	// Some constants defined in the description.
	double h_boundaries = 100.0f;
	double h_inErr = 1e-15;

	// Alpha constant in formula.
	double h_alpha;

	// Read input
	fscanf(stdin, "%lf", &h_deltaT);
	fscanf(stdin, "%lf", &h_deltaH);
	fscanf(stdin, "%lf", &h_sizeZ);
	fscanf(stdin, "%lf", &h_sizeY);
	fscanf(stdin, "%lf", &h_sizeX);

	// Calculate ne number of elements in x, y and z axis.
	unsigned int h_npX = (unsigned int) (h_sizeX / h_deltaH);
	unsigned int h_npY = (unsigned int) (h_sizeY / h_deltaH);
	unsigned int h_npZ = (unsigned int) (h_sizeZ / h_deltaH);

	printf("npZ = %d, npY = %d, npX = %d\n", h_npZ, h_npY, h_npX);

	h_alpha = h_deltaT / (h_deltaH * h_deltaH);

	// Call the device to calculate the heating.
	onDevice(h_npX, h_npY, h_npZ, h_deltaH, h_deltaT, h_alpha, h_boundaries, h_inErr);
	// mdf_heat(h_u0, h_u1, h_npX, h_npY, h_npZ, h_deltaH, h_deltaT, 1e-15, 100.0f);
	//mdf_print(u1,  npX, npY, npZ);

	return EXIT_SUCCESS;
}

int main (int argc, char *argv[]){
	return onHost();
}
